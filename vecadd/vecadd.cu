#include <stdio.h>
#include <hip/hip_runtime.h>

//very basic version of vec add, each thread do one adding.
__global__ void vecadd(const float *A, const float *B, float *C, int numEle){
    int  idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx < numEle ) {
        C[idx] = A[idx] + B[idx];
    }
}


//host main routine
int main(void){
    hipError_t err = hipSuccess;
    int numEle = 50000;
    size_t size = numEle * sizeof(float);
    //host vec
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    for (int i = 0; i< numEle; ++i){
        h_A[i] = rand()/(float) RAND_MAX;
        h_B[i] = rand()/(float) RAND_MAX;
    }
    //device vec
    float *d_A = NULL;
    err = hipMalloc((void **) &d_A, size);

    if(err != hipSuccess){
        fprintf(stderr, "fail to allocate device vec A (errorcode %s )!\n", hipGetErrorString(err)); exit(EXIT_FAILURE);
    }

    float *d_B = NULL;
    err = hipMalloc((void **) &d_B, size);
    float *d_C = NULL;
    err = hipMalloc((void **) &d_C, size);
    //mem copy to device.
    hipMemcpy(d_A,h_A,size, hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size, hipMemcpyHostToDevice);
    hipMemcpy(d_C,h_C,size, hipMemcpyHostToDevice);
    //Launch the vec add Kernel
    int threadsperblock = 256;
    int blockspergrid = (numEle + threadsperblock - 1)/threadsperblock;
    
    vecadd<<<blockspergrid,threadsperblock>>>(d_A,d_B,d_C,numEle);
    err = hipGetLastError();

    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
    //verifying
    for (int i = 0; i < numEle; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    
} 
